#include "hip/hip_runtime.h"
/*  runmcmc.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Isis
    Innovation Limited ("Isis"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    innovation@isis.ox.ac.uk quoting reference DE/9564. */

#include "xfibresoptions.h"
#include <hiprand.h>
#include "runmcmc_kernels.cu"
#include "sync_check.h"

#include <host_vector.h>
#include <device_vector.h> 

#include <time.h>
#include <sys/time.h>
#include "init_gpu.h"

using namespace Xfibres;

////////////////////////////////////////////////////// 
//   MCMC IN GPU
////////////////////////////////////////////////////// 

void init_Fibres_Multifibres(	//INPUT
				thrust::device_vector<float> 			datam_gpu,
				thrust::device_vector<float> 			params_gpu,
				thrust::device_vector<float> 			tau_gpu,
				thrust::device_vector<float> 			bvals_gpu,
				thrust::device_vector<double> 			alpha_gpu,
				thrust::device_vector<double> 			beta_gpu,
				const int 					ndirections,
				string 						output_file, 
				//OUTPUT
				thrust::device_vector<FibreGPU>& 		fibres_gpu,
				thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
				thrust::device_vector<double>&			signals_gpu,
				thrust::device_vector<double>&			isosignals_gpu)
{
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app );
   	myfile << "----- MCMC ALGORITHM PART INITIALITATION IN GPU ----- " << "\n";  	

   	struct timeval t1,t2;
   	double time;
   	gettimeofday(&t1,NULL);

	int nvox = multifibres_gpu.size();

	xfibresOptions& opts = xfibresOptions::getInstance();
	int nfib= opts.nfibres.value();
	int nparams_fit = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()>=2) nparams_fit++;
	if(opts.f0.value()) nparams_fit++;

	thrust::device_vector<double> angtmp_gpu;
	angtmp_gpu.resize(nvox*ndirections*nfib);
	

	bool gradnonlin = opts.grad_file.set();

	int blocks = nvox; 
  	dim3 Dim_Grid_MCMC(blocks, 1);
  	dim3 Dim_Block_MCMC(THREADS_BLOCK_MCMC ,1);	///dimensions for MCMC

	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *params_ptr = thrust::raw_pointer_cast(params_gpu.data());	
	float *tau_ptr = thrust::raw_pointer_cast(tau_gpu.data());	
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());
	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());

	int amount_shared = (THREADS_BLOCK_MCMC)*sizeof(double) + (3*nfib + 9)*sizeof(float) + sizeof(int);

	myfile << "Shared Memory Used in init_Fibres_Multifibres: " << amount_shared << "\n";

	init_Fibres_Multifibres_kernel<<< Dim_Grid_MCMC, Dim_Block_MCMC, amount_shared>>>(datam_ptr, params_ptr, tau_ptr, bvals_ptr, alpha_ptr, beta_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams_fit, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.rician.value(), opts.ardf0.value(), opts.all_ard.value(), opts.no_ard.value(), gradnonlin, angtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr);
	sync_check("init_Fibres_Multifibres_kernel");

	gettimeofday(&t2,NULL);
    	time=timeval_diff(&t2,&t1);
   	myfile << "TIME TOTAL: " << time << " seconds\n"; 
	myfile << "-----------------------------------------------------" << "\n\n" ; 
	myfile.close();
}

void runmcmc_burnin(	//INPUT
			thrust::device_vector<float> 			datam_gpu,
			thrust::device_vector<float> 			bvals_gpu,
			thrust::device_vector<double> 			alpha_gpu,
			thrust::device_vector<double> 			beta_gpu,
			const int 					ndirections,
			double 						seed,
			string 						output_file, 
			//INPUT-OUTPUT
			thrust::device_vector<FibreGPU>& 		fibres_gpu,
			thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
			thrust::device_vector<double>&			signals_gpu,
			thrust::device_vector<double>&			isosignals_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app ); 
   	myfile << "--------- MCMC ALGORITHM PART BURNIN IN GPU --------- " << "\n";  	

   	struct timeval t1,t2,t_tot1,t_tot2;
   	double time,timecurand,timemcmc;
   	time=0;
   	timecurand=0;
   	timemcmc=0;

   	gettimeofday(&t_tot1,NULL);

   	size_t free,total;
	
	int nvox = multifibres_gpu.size();
   	int nfib= opts.nfibres.value();
	int nparams;

	bool gradnonlin=opts.grad_file.set();

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;	
	if(opts.modelnum.value()>=2) nparams++;
	if(opts.modelnum.value()==3) nparams++;	
	if(opts.rician.value()) nparams++;

	thrust::device_vector<float> recors_null_gpu;
	recors_null_gpu.resize(1);

	thrust::device_vector<double> angtmp_gpu;
	thrust::device_vector<double> oldangtmp_gpu;
	thrust::device_vector<double> oldsignals_gpu;
	thrust::device_vector<double> oldisosignals_gpu;
	
	angtmp_gpu.resize(nvox*ndirections*nfib);
	oldangtmp_gpu.resize(nvox*ndirections);
	oldsignals_gpu.resize(nvox*ndirections*nfib);
	oldisosignals_gpu.resize(nvox*ndirections);
   
   	unsigned int totalrandoms=(opts.nburn.value() * nvox * nparams);

   	hipMemGetInfo(&free,&total);
   	myfile << "Free memory Before Randoms: "<< free <<  " ---- Total memory: " << total << "\n";
   	//4 bytes each float, 2 random arrays, and 80% of total memory at this moment 
   	unsigned int maxrandoms=((free*0.8)/(4*2)); 

   	myfile << "Total randoms: " << totalrandoms << "\n"; 
   	myfile << "Max randoms: " << maxrandoms << "\n"; 
   
   	int steps; //num iter if not enough memory
   	int minrandoms; //min num of randoms ensamble
   	minrandoms= nvox * nparams;

   	int iters_step=0;
	int nrandoms=0;	

   	if(totalrandoms>maxrandoms){ 
		iters_step = maxrandoms / minrandoms; 			//iterations in each step
		nrandoms = iters_step*minrandoms;			//nrandoms for each step
		steps =  (opts.nburn.value()/iters_step);  		//repeat process steps times, no enough memory for all randoms 			
   	}else{ 
		nrandoms = totalrandoms;
		iters_step= opts.nburn.value();
		steps = 0;
  	}
	if(nrandoms%2){							//CURAND must generates multiples of 2 randoms
		nrandoms++;
	}
	
   	myfile << "Process " << opts.nburn.value() << " iterations divided in "<< steps << " steps with "<< iters_step << " iterations in each one" << "\n";    

   	int last_step = opts.nburn.value() - (iters_step*steps);
   	int last_randoms = (last_step*minrandoms);
	if(last_randoms%2){						//CURAND must generates multiples of 2 randoms
		last_randoms++;
	}

   	myfile << "Last step with " << last_step << " iterations" << "\n"; 
	
	thrust::device_vector<float> randomsN_gpu;
	thrust::device_vector<float> randomsU_gpu;	
	randomsN_gpu.resize(nrandoms);
	randomsU_gpu.resize(nrandoms);

   	hipMemGetInfo(&free,&total);
   	myfile << "Free memory after Malloc Randoms: "<< free <<  " ---- Total memory: " << total << "\n";
   
  	int blocks = nvox;        
  	dim3 Dim_Grid(blocks, 1);
  	dim3 Dim_Block(THREADS_BLOCK_MCMC,1);	//dimensions for MCMC   

   	myfile << "\n" << "NUM BLOCKS: " << blocks << "\n"; 
   	myfile << "THREADS PER BLOCK : " << THREADS_BLOCK_MCMC << "\n\n"; 	

   	hiprandGenerator_t gen;
   	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
   	hiprandSetPseudoRandomGeneratorSeed(gen,seed);

	//get pointers
	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	float *randomsN_ptr = thrust::raw_pointer_cast(randomsN_gpu.data());
	float *randomsU_ptr = thrust::raw_pointer_cast(randomsU_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());

	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	double *oldangtmp_ptr = thrust::raw_pointer_cast(oldangtmp_gpu.data());
	double *oldsignals_ptr = thrust::raw_pointer_cast(oldsignals_gpu.data());
	double *oldisosignals_ptr = thrust::raw_pointer_cast(oldisosignals_gpu.data());

	float *records_null = thrust::raw_pointer_cast(recors_null_gpu.data());

	int amount_shared = (THREADS_BLOCK_MCMC)*sizeof(double) + (10*nfib + 2*nparams + 27)*sizeof(float) + (7*nfib + 21)*sizeof(int);

	myfile << "Shared Memory Used in runmcmc_burnin: " << amount_shared << "\n";
	
   	for(int i=0;i<steps;i++){

   		gettimeofday(&t1,NULL);

	   	hiprandStatus_t status = hiprandGenerateNormal(gen,randomsN_ptr,nrandoms,0,1); 
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}
	   	status = hiprandGenerateUniform(gen,randomsU_ptr,nrandoms);	//generate randoms
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}

 	   	gettimeofday(&t2,NULL);
    	   	timecurand+=timeval_diff(&t2,&t1);

	   	gettimeofday(&t1,NULL);

	   	runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr, randomsN_ptr, randomsU_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), iters_step, (i*iters_step), 0, 0, 0, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr,records_null,records_null,records_null,records_null,records_null,records_null,records_null,records_null, records_null);   
	   	sync_check("runmcmc_burnin_kernel");

 	   	gettimeofday(&t2,NULL);
    	   	timemcmc+=timeval_diff(&t2,&t1);
   	}

   	gettimeofday(&t1,NULL); 

   	if(nvox!=0){
   		hiprandStatus_t status = hiprandGenerateNormal(gen,randomsN_ptr,last_randoms,0,1);
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}
   		status = hiprandGenerateUniform(gen,randomsU_ptr,last_randoms); 	//generate randoms
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}
   	}
	
   	gettimeofday(&t2,NULL);
   	timecurand+=timeval_diff(&t2,&t1);

   	gettimeofday(&t1,NULL);

   	if(nvox!=0){
		runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr, randomsN_ptr, randomsU_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), last_step, (steps*iters_step), 0, 0, 0, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr,records_null,records_null,records_null,records_null,records_null,records_null,records_null, records_null,records_null); 
   		sync_check("runmcmc_burnin_kernel");
   	}

   	gettimeofday(&t2,NULL);
   	timemcmc+=timeval_diff(&t2,&t1);

    	myfile << "TIME CURAND: " << timecurand << " seconds\n"; 
    	myfile << "TIME RUNMCMC: " << timemcmc << " seconds\n"; 
   
   	hiprandDestroyGenerator(gen);

	gettimeofday(&t_tot2,NULL);
    	time=timeval_diff(&t_tot2,&t_tot1);
   	myfile << "TIME TOTAL: " << time << " seconds\n"; 
	myfile << "-----------------------------------------------------" << "\n\n" ; 
	myfile.close();

   	sync_check("runmcmc_burnin");
}


void runmcmc_record(	//INPUT
			thrust::device_vector<float> 			datam_gpu,
			thrust::device_vector<float> 			bvals_gpu,
			thrust::device_vector<double> 			alpha_gpu,
			thrust::device_vector<double> 			beta_gpu,
			thrust::device_vector<FibreGPU> 		fibres_gpu,
			thrust::device_vector<MultifibreGPU> 		multifibres_gpu,
			thrust::device_vector<double>			signals_gpu,
			thrust::device_vector<double>			isosignals_gpu,
			const int 					ndirections,
			double 						seed,
			string 						output_file, 
			//OUTPUT
			thrust::device_vector<float>&			rf0_gpu,
			thrust::device_vector<float>&			rtau_gpu,
			thrust::device_vector<float>&			rs0_gpu,
			thrust::device_vector<float>&			rd_gpu,
			thrust::device_vector<float>&			rdstd_gpu,
			thrust::device_vector<float>&			rR_gpu,
			thrust::device_vector<float>&			rth_gpu,
			thrust::device_vector<float>&			rph_gpu,
			thrust::device_vector<float>&			rf_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app );
   	myfile << "--------- MCMC ALGORITHM PART RECORD IN GPU --------- " << "\n"; 	

   	struct timeval t1,t2,t_tot1,t_tot2;
   	double time,timecurand,timemcmc;
   	time=0;
   	timecurand=0;
   	timemcmc=0;

   	gettimeofday(&t_tot1,NULL);

   	size_t free,total;

	int totalrecords = (opts.njumps.value()/opts.sampleevery.value()); 
	
	int nvox = multifibres_gpu.size();
   	int nfib= opts.nfibres.value();
	int nparams;

	bool gradnonlin=opts.grad_file.set();

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;	
	if(opts.modelnum.value()>=2) nparams++;
	if(opts.modelnum.value()==3) nparams++;	
	if(opts.rician.value()) nparams++;

	thrust::device_vector<double> angtmp_gpu;
	thrust::device_vector<double> oldangtmp_gpu;
	thrust::device_vector<double> oldsignals_gpu;
	thrust::device_vector<double> oldisosignals_gpu;
	
	angtmp_gpu.resize(nvox*ndirections*nfib);
	oldangtmp_gpu.resize(nvox*ndirections);
	oldsignals_gpu.resize(nvox*ndirections*nfib);
	oldisosignals_gpu.resize(nvox*ndirections);
   
   	unsigned int totalrandoms=(opts.njumps.value() * nvox * nparams);

   	hipMemGetInfo(&free,&total);
   	myfile << "Free memory Before Randoms: "<< free <<  " ---- Total memory: " << total << "\n";
   	//4 bytes each float, 2 random arrays, and 80% of total memory at this moment 
   	unsigned int maxrandoms=((free*0.8)/(4*2)); 

   	myfile << "Total randoms: " << totalrandoms << "\n"; 
   	myfile << "Max randoms: " << maxrandoms << "\n"; 
   
   	int steps; //num iter if not enough memory
   	int minrandoms; //min num of randoms ensamble
   	minrandoms= nvox * nparams;

   	int iters_step=0;
	int nrandoms=0;	

   	if(totalrandoms>maxrandoms){ 
		iters_step = maxrandoms / minrandoms; 		//iterations in each step
		nrandoms = iters_step*minrandoms;		//nrandoms for each step
		steps =  (opts.njumps.value()/iters_step);  	//repeat process steps times, no enough memory for all randoms 			
   	}else{ 
		nrandoms = totalrandoms;
		iters_step= opts.njumps.value();
		steps = 0;
  	}   
	if(nrandoms%2){						//CURAND must generates multiples of 2 randoms
		nrandoms++;
	}

   	myfile << "Process " << opts.njumps.value() << " iterations divided in "<< steps << " steps with "<< iters_step << " iterations in each one" << "\n";    

   	int last_step = opts.njumps.value() - (iters_step*steps);
   	int last_randoms = (last_step*minrandoms); 
	if(last_randoms%2){					//CURAND must generates multiples of 2 randoms
		last_randoms++;
	}

   	myfile << "Last step with " << last_step << " iterations" << "\n"; 
	
	thrust::device_vector<float> randomsN_gpu;
	thrust::device_vector<float> randomsU_gpu;	
	randomsN_gpu.resize(nrandoms);
	randomsU_gpu.resize(nrandoms);

   	hipMemGetInfo(&free,&total);
   	myfile << "Free memory after Malloc Randoms: "<< free <<  " ---- Total memory: " << total << "\n";
   
  	int blocks = nvox;        
  	dim3 Dim_Grid(blocks, 1);
  	dim3 Dim_Block(THREADS_BLOCK_MCMC,1);	//dimensions for MCMC   

   	myfile << "\n" << "NUM BLOCKS: " << blocks << "\n"; 
   	myfile << "THREADS PER BLOCK : " << THREADS_BLOCK_MCMC << "\n\n"; 	

   	hiprandGenerator_t gen;
   	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
   	hiprandSetPseudoRandomGeneratorSeed(gen,seed);

	//get pointers
	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	float *randomsN_ptr = thrust::raw_pointer_cast(randomsN_gpu.data());
	float *randomsU_ptr = thrust::raw_pointer_cast(randomsU_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());

	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	double *oldangtmp_ptr = thrust::raw_pointer_cast(oldangtmp_gpu.data());
	double *oldsignals_ptr = thrust::raw_pointer_cast(oldsignals_gpu.data());
	double *oldisosignals_ptr = thrust::raw_pointer_cast(oldisosignals_gpu.data());
	
	float *rf0_ptr = thrust::raw_pointer_cast(rf0_gpu.data());
	float *rtau_ptr = thrust::raw_pointer_cast(rtau_gpu.data());
	float *rs0_ptr = thrust::raw_pointer_cast(rs0_gpu.data());
	float *rd_ptr = thrust::raw_pointer_cast(rd_gpu.data());
	float *rdstd_ptr = thrust::raw_pointer_cast(rdstd_gpu.data());
	float *rR_ptr = thrust::raw_pointer_cast(rR_gpu.data());	
	float *rth_ptr = thrust::raw_pointer_cast(rth_gpu.data());
	float *rph_ptr = thrust::raw_pointer_cast(rph_gpu.data());
	float *rf_ptr = thrust::raw_pointer_cast(rf_gpu.data());

	int amount_shared = (THREADS_BLOCK_MCMC)*sizeof(double) + (10*nfib + 2*nparams + 27)*sizeof(float) + (7*nfib + 21)*sizeof(int);

	myfile << "Shared Memory Used in runmcmc_record: " << amount_shared << "\n";

   	for(int i=0;i<steps;i++){

   		gettimeofday(&t1,NULL);

	   	hiprandStatus_t status = hiprandGenerateNormal(gen,randomsN_ptr,nrandoms,0,1);
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}
	   	status = hiprandGenerateUniform(gen,randomsU_ptr,nrandoms);	//generate randoms
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}

 	   	gettimeofday(&t2,NULL);
    	   	timecurand+=timeval_diff(&t2,&t1);

	   	gettimeofday(&t1,NULL);

	   	runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr, randomsN_ptr, randomsU_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), iters_step, (i*iters_step), opts.nburn.value(), opts.sampleevery.value(), totalrecords, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr, rf0_ptr, rtau_ptr, rs0_ptr, rd_ptr, rdstd_ptr, rR_ptr, rth_ptr, rph_ptr, rf_ptr);
	   	sync_check("runmcmc_record_kernel");

 	   	gettimeofday(&t2,NULL);
    	   	timemcmc+=timeval_diff(&t2,&t1);
   	}

   	gettimeofday(&t1,NULL);

   	if(nvox!=0){
   		hiprandStatus_t status = hiprandGenerateNormal(gen,randomsN_ptr,last_randoms,0,1);
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}
   		status = hiprandGenerateUniform(gen,randomsU_ptr,last_randoms); 	//generate randoms
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("Failure generating cuda random numbers: %d\n",status);
			exit(1);
		}
   	}
	
   	gettimeofday(&t2,NULL);
   	timecurand+=timeval_diff(&t2,&t1);

   	gettimeofday(&t1,NULL);

   	if(nvox!=0){
		runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr,randomsN_ptr, randomsU_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), last_step, (steps*iters_step), opts.nburn.value(), opts.sampleevery.value(), totalrecords, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr, rf0_ptr, rtau_ptr, rs0_ptr, rd_ptr, rdstd_ptr, rR_ptr, rth_ptr, rph_ptr, rf_ptr);   
   		sync_check("runmcmc_record_kernel");
   	}

   	gettimeofday(&t2,NULL);
   	timemcmc+=timeval_diff(&t2,&t1);


    	myfile << "TIME CURAND: " << timecurand << " seconds\n"; 
    	myfile << "TIME RUNMCMC: " << timemcmc << " seconds\n"; 
   
   	hiprandDestroyGenerator(gen);

	gettimeofday(&t_tot2,NULL);
    	time=timeval_diff(&t_tot2,&t_tot1);
   	myfile << "TIME TOTAL: " << time << " seconds\n"; 
	myfile << "-----------------------------------------------------" << "\n" ;
	myfile.close(); 
	
   	sync_check("runmcmc_record");
}
